#include <iostream>
#include <string>
#include <time.h>
#include <chrono>
#include <cxxopts.hpp>
#include <iomanip>
#include <sstream>

#include "compute.cuh"

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 256
#endif

enum class Command {
    Account,
    Contract,
    Help
};

struct Options {
    std::string prefix;
    std::string suffix;
    bool zeroBytes;
    bool ignoreCase;
    Command command;
};

Command parseCommand(const std::string& commandStr) {
    if (commandStr == "account") {
        return Command::Account;
    } else if (commandStr == "contract") {
        return Command::Contract;
    } else if (commandStr == "help") {
        return Command::Help;
    } else {
        return Command::Help;
    }
}

std::pair<Options, cxxopts::Options> parseOptions(int argc, char* argv[]) {
    Options options;

    cxxopts::Options cmdlineOptions("ethereum_vanity", "Bruteforce Ethereum addresses");

    cmdlineOptions.add_options()
        ("p,prefix", "Address prefix", cxxopts::value<std::string>(options.prefix))
        ("s,suffix", "Address suffix", cxxopts::value<std::string>(options.suffix))
        ("z,zero-bytes", "Bruteforce forever until stopped by the user, keeping the address with the most zero bytes", cxxopts::value<bool>(options.zeroBytes))
        ("i,ignore-case", "Ignore case for prefix and suffix", cxxopts::value<bool>(options.ignoreCase))
        ("h,help", "Print help")
        ("V,version", "Print version");

    cmdlineOptions.parse_positional({"command"});

    try {
        auto result = cmdlineOptions.parse(argc, argv);

        if (result.count("help")) {
            throw std::runtime_error("Print help");
        }

        if (result.count("version")) {
            throw std::runtime_error("Print version");
        }

        if (result.count("command") == 0) {
            throw std::runtime_error("No command specified");
        }

        std::string commandStr = result["command"].as<std::string>();
        options.command = parseCommand(commandStr);
    } catch (const std::exception& e) {
        std::cerr << "Error parsing command line options: " << e.what() << std::endl;
        std::cerr << cmdlineOptions.help() << std::endl;
        exit(1);
    }

    return {options, cmdlineOptions};
}

void printHelp(const cxxopts::Options& options) {
    std::cout << options.help() << std::endl;
}

int main(int argc, char* argv[]) {
    auto [options, cmdlineOptions] = parseOptions(argc, argv);

    switch (options.command) {
        case Command::Account: {
            std::cout << "Bruteforce a private key" << std::endl;
            const int privateKeySize = 32;
            uint8_t privateKey[privateKeySize];

            // Allocation de la mémoire sur le GPU
            uint8_t* dev_privateKey;
            hipMalloc((void**)&dev_privateKey, privateKeySize * sizeof(uint8_t));

            // Définition de la configuration des blocs et des threads
            dim3 blockDim(THREADS_PER_BLOCK); // Nombre de threads par bloc
            dim3 gridDim(1); // Nombre de blocs

            // Exécution du kernel CUDA
            generatePrivateKey<<<gridDim, blockDim>>>(dev_privateKey);

            // Copie du résultat depuis le GPU vers le CPU
            hipMemcpy(privateKey, dev_privateKey, privateKeySize * sizeof(uint8_t), hipMemcpyDeviceToHost);

            // Libération de la mémoire sur le GPU
            hipFree(dev_privateKey);

            // Conversion du privateKey en hexadécimal
            std::stringstream ss;
            for (int i = 0; i < privateKeySize; i++) {
                ss << std::setw(2) << std::setfill('0') << std::hex << static_cast<int>(privateKey[i]);
            }
            std::string privateKeyHex = ss.str();

            std::cout << "Private Key: " << privateKeyHex << std::endl;

            // Vérification de l'adresse Ethereum
            const std::string prefix = "0x"; // Préfixe de l'adresse Ethereum à rechercher
            const int prefixSize = prefix.size() / 2; // Taille du préfixe en octets

            const int numKeys = 1; // Nombre de clés privées à vérifier
            bool results[numKeys]; // Tableau pour stocker les résultats de vérification

            checkAddresses(privateKey, numKeys, reinterpret_cast<const uint8_t*>(prefix.data()), prefixSize, results);

            // Vérification du résultat
            if (results[0]) {
                std::cout << "Address with prefix " << prefix << " found!" << std::endl;
            } else {
                std::cout << "No address with prefix " << prefix << " found." << std::endl;
            }

            break;
        }

        case Command::Contract: {
            // Code pour la commande 'contract'
            std::cout << "Bruteforce a CREATE2 salt" << std::endl;
            // Ajoutez le code pour la commande 'contract' ici
            break;
        }
        case Command::Help: {
            printHelp(cmdlineOptions);
            break;
        }
    }

    return 0;
}
