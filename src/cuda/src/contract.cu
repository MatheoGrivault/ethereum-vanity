#include "hip/hip_runtime.h"
#include "include/config.hpp"
#include "include/keccak.cuh"

#ifndef Contract_Thread
#define Contract_Thread 512
#endif

__global__ void generatesalt(hiprandState* states, unsigned char* salts) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init((unsigned long long) clock() + index, 0, 0, &states[index]);

    // Générer 32 octets (256 bits) de sel
    for (int i = 0; i < 32; i++) {
        salts[index * 32 + i] = (unsigned char)(hiprand(&states[index]) % 256);
    }
}


__device__ void keccak_hash_compute(BYTE* in, WORD inlen, BYTE* out, WORD n_outbit, WORD n_batch) {
    // In-memory allocation since we are on device side
    const WORD KECCAK_BLOCK_SIZE = (n_outbit >> 3);

    // Iterate over each batch
    for (int i = 0; i < n_batch; ++i) {
        BYTE* cuda_in = in + i * inlen;
        BYTE* cuda_out = out + i * KECCAK_BLOCK_SIZE;
        
        device_keccak_hash(cuda_in, inlen, cuda_out, KECCAK_BLOCK_SIZE);
    }
}   

__global__ void computeContractAdresse(unsigned char* salts, unsigned char* deploymentAddress, size_t deploymentAddressLen, unsigned char* bytecode, size_t bytecodeLen, unsigned char* contractAddresses) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    unsigned char salt[32];
    for (int i = 0; i < 32; i++) {
        salt[i] = salts[index * 32 + i];
    }

    unsigned char data[132];  // valeur maximale connue pour deploymentAddressLen + 32
    memcpy(data, deploymentAddress, deploymentAddressLen);
    memcpy(data + deploymentAddressLen, salt, 32);

    unsigned char hash[32];
    keccak_hash_compute(data, deploymentAddressLen + 32, hash, 256, 1);

    memcpy(contractAddresses + (index * 20), hash + 12, 20);
}


__device__ int cuda_memcmp(const void* s1, const void* s2, size_t n) {
    const unsigned char *p1 = (unsigned char*)s1, *p2 = (unsigned char*)s2;

    while(n--) {
        if( *p1 != *p2 ) {
            return *p1 - *p2;
        } else {
            p1++;
            p2++;
        }
    }

    return 0;
}

__device__ bool verifyPrefixAndSuffix(unsigned char* address, unsigned char* prefix, size_t prefixLen, unsigned char* suffix, size_t suffixLen) {
    if (prefixLen > 0 && cuda_memcmp(address, prefix, prefixLen) != 0) {
        return false;
    }

    if (suffixLen > 0 && cuda_memcmp(address + 20 - suffixLen, suffix, suffixLen) != 0) {
        return false;
    }

    return true;
}

__device__ int calculateNumZeroBytes(unsigned char* address) {
    int zeroCount = 0;

    for (int i = 0; i < 20; i++) {
        if (address[i] == 0) {
            zeroCount++;
        } else {
            break;
        }
    }

    return zeroCount;
}

__device__ void generateContractAddress(const unsigned char* deploymentAddress, size_t deploymentAddressLen,
                                       const unsigned char* bytecode, size_t bytecodeLen,
                                       const unsigned char* salt, unsigned char* contractAddress)  {
    uint8_t* data = new uint8_t[deploymentAddressLen + 32 + bytecodeLen]; // La taille du sel est de 32 octets
    memcpy(data, deploymentAddress, deploymentAddressLen);
    memcpy(data + deploymentAddressLen, salt, 32);
    memcpy(data + deploymentAddressLen + 32, bytecode, bytecodeLen);

    uint8_t* hash = new uint8_t[32];
    keccak_hash_compute(data, deploymentAddressLen + 32 + bytecodeLen, hash, 256, 1);

    memcpy(contractAddress, hash + 12, 20);

    delete[] data;
    delete[] hash;
}


__global__ void verifyContractAdresse(const unsigned char* deploymentAddress, size_t deploymentAddressLen,
                                        const unsigned char* bytecode, size_t bytecodeLen,
                                        const unsigned char* salts, size_t numSalts,
                                        unsigned char* validAddresses, int* validAddressesCount,
                                        unsigned char* prefix, size_t prefixLen,
                                        unsigned char* suffix, size_t suffixLen){
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < numSalts) {
        uint8_t contractAddress[20]; // Allocation sur la pile plutôt que dynamiquement
        generateContractAddress(deploymentAddress, deploymentAddressLen, bytecode, bytecodeLen, salts + index * 32, contractAddress);

        if (verifyPrefixAndSuffix(contractAddress, prefix, prefixLen, suffix, suffixLen)) {
            int nZeroBytes = calculateNumZeroBytes(contractAddress);

            // Utilisez des variables locales pour réduire les accès atomiques et la concurrence
            int localMax = atomicMax(validAddressesCount, nZeroBytes);

            // Assurez-vous que cette condition est correctement synchronisée pour éviter les conditions de concurrence.
            if (nZeroBytes > localMax) { 
                for (int i = 0; i < 20; ++i) { // Remplacer memcpy par une boucle
                    validAddresses[i] = contractAddress[i];
                }
            }
        }
    }
}

