#include "include/config.hpp"

int THREADS_PER_BLOCK;
#define TRYTES_PER_TRIT 50000

enum class Command {    
    Account,
    Contract,
    Help
};

struct Options {
    std::string prefix;
    std::string suffix;
    std::string deployerAddress;
    std::string Bytecode;
    bool zeroBytes;
    bool ignoreCase;
    bool loop;
    int loopCount;
    Command command;
};

Command parseCommand(const std::string& commandStr) {
    if (commandStr == "account") {
        return Command::Account;
    } else if (commandStr == "contract") {
        return Command::Contract;
    } else {
        throw CLI::ValidationError("Invalid command. Valid commands are: account, contract");
    }
}

Options parseOptions(int argc, char* argv[]) {
    Options options;
    options.loopCount=1;

    CLI::App app{"Bruteforce Ethereum addresses"};

    CLI::App *account_subcommand = app.add_subcommand("account", "Bruteforce a private key");
    CLI::App *contract_subcommand = app.add_subcommand("contract", "Bruteforce a CREATE2 salt");

    //account command options
    account_subcommand->callback([&]() { options.command = Command::Account; });
    account_subcommand->add_option("-p,--prefix", options.prefix, "Address prefix");
    account_subcommand->add_option("-s,--suffix", options.suffix, "Address suffix");
    account_subcommand->add_flag("-z,--zero-bytes", options.zeroBytes, "Bruteforce forever until stopped by the user, keeping the address with the most zero bytes");
    account_subcommand->add_flag("-i,--ignore-case", options.ignoreCase, "Ignore case for prefix and suffix");
    account_subcommand->add_flag("-l,--loop", options.loop, "Loop through all private keys");

    //contract command options
    contract_subcommand->callback([&]() { options.command = Command::Contract; });
    contract_subcommand->add_option("-p,--prefix", options.prefix, "Contract address prefix");
    contract_subcommand->add_option("-s,--suffix", options.suffix, "Contract address suffix");
    contract_subcommand->add_option("-d,--deployer", options.deployerAddress, "Deployer's address");
    contract_subcommand->add_option("-b,--bytecode", options.Bytecode, "Contract's bytecode");
    contract_subcommand->add_flag("-z,--zero-bytes", options.zeroBytes, "Bruteforce forever until stopped by the user, keeping the address with the most zero bytes");
    contract_subcommand->add_flag("-i,--ignore-case", options.ignoreCase, "Ignore case for prefix and suffix");
    contract_subcommand->add_flag("-l,--loop", options.loop, "Loop through all salts");

    app.add_flag_function("-v,--version", [](int) { std::cout << "Version 1.0.0" << std::endl; exit(0); }, "Print version and exit");

    try {
        app.parse(argc, argv);
    } catch (const CLI::ParseError &e) {
        std::cerr << "Error parsing command line options: " << e.what() << std::endl;
        exit(app.exit(e));
    }

    return options;
}

bool userprompt(){
    char decision;
    
    std::cout << "Do you want to continue ? (y/n)" << std::endl;
    std::cin >> decision;
    if (decision == 'y' || decision == 'Y') {
        return true;
    } else {
        return false;
    }
}

void checkCudaError(hipError_t error, const char* functionName) {
    if (error != hipSuccess) {
        std::cerr << "CUDA Error (" << functionName << "): " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE); // Quitte le programme en cas d'erreur
    }
}


int main(int argc, char* argv[]) {

    int deviceCount;
    hipGetDeviceCount(&deviceCount);   

    int device(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    THREADS_PER_BLOCK = maxThreadsPerBlock;

    Options options = parseOptions(argc, argv);
    std::cout << "Start using " << THREADS_PER_BLOCK << " threads per block\n" << std::endl;
    switch (options.command) {
        case Command::Account: {
            std::cout << "Bruteforce a private key\n" << std::endl;

            const int numKeys = THREADS_PER_BLOCK;
            unsigned char* privateKeys = new unsigned char[numKeys * 32];
            unsigned char* dev_privateKeys;
            hipMalloc((void**)&dev_privateKeys, numKeys * 32 * sizeof(unsigned char));

            unsigned char* dev_validPrivateKeys;
            int* dev_validCount;
            hipMalloc((void**)&dev_validPrivateKeys, numKeys * 32 * sizeof(unsigned char));
            hipMalloc((void**)&dev_validCount, sizeof(int));

            unsigned char* validPrivateKeys = new unsigned char[numKeys * 32];
            int* validCount = new int(0);

            dim3 blockDim(THREADS_PER_BLOCK);
            dim3 gridDim((numKeys + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

            while (true) {
                generatePrivateKey<<<gridDim, blockDim>>>(dev_privateKeys, numKeys);
                hipDeviceSynchronize();
                unsigned char* host_privateKeys = new unsigned char[numKeys * 32];
                hipMemcpy(host_privateKeys, dev_privateKeys, numKeys * 32 * sizeof(unsigned char), hipMemcpyDeviceToHost);

                hipError_t error = hipGetLastError();
                if (error != hipSuccess) {
                    std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
                    break;  // Sortir de la boucle en cas d'erreur CUDA
                }

                bool* results = new bool[numKeys];
                std::string* result_addresses = new std::string[numKeys];

                checkAddresses(privateKeys, numKeys, options.prefix.c_str(), options.suffix.c_str(), results, result_addresses);

                for (int i = 0; i < numKeys; ++i) {
                    if (results[i]) {
                        std::cout << "Address with prefix " << options.prefix << " and suffix " << options.suffix << " found for private key " << i << "!" << std::endl;
                        std::cout << "Address: " << result_addresses[i] << std::endl;
                    }
                }

                delete[] results;
                delete[] result_addresses;

                options.loopCount += 1;
                if (options.loop == true) {
                    if (options.loopCount % TRYTES_PER_TRIT == 0) {
                        std::cout << "Loop number: " << options.loopCount << std::endl;
                        bool decision = userprompt();
                        if (decision == true) {
                            continue;
                        } else {
                            break;
                        }
                    }
                } else {
                    break;
                }
            }

            // Libération de la mémoire à la fin
            delete[] privateKeys;
            delete[] validPrivateKeys;
            delete validCount;
            hipFree(dev_privateKeys);
            hipFree(dev_validPrivateKeys);
            hipFree(dev_validCount);

            break;
        }


        case Command::Contract: {
            std::cout << "Bruteforce a CREATE2 salt\n" << std::endl;

            const int numSalts = THREADS_PER_BLOCK;
            const int saltSize = 32;
            unsigned char* h_salts = new unsigned char[numSalts * saltSize];

            while (true) {
                
                unsigned char* dev_salts;
                hiprandState* devStates;

                hipMalloc((void**)&devStates, numSalts * sizeof(hiprandState));
                checkCudaError(hipGetLastError(), "hipMalloc for devStates");
                hipMalloc((void**)&dev_salts, numSalts * saltSize);
                checkCudaError(hipGetLastError(), "hipMalloc for dev_salts");

                dim3 blockDim(THREADS_PER_BLOCK);
                dim3 gridDim((numSalts + blockDim.x - 1) / blockDim.x);

                generatesalt<<<gridDim, blockDim>>>(devStates, dev_salts);
                checkCudaError(hipGetLastError(), "generatesalt kernel launch");

                hipMemcpy(h_salts, dev_salts, numSalts * saltSize, hipMemcpyDeviceToHost);
                checkCudaError(hipGetLastError(), "hipMemcpy from dev_salts to h_salts");

                std::string deploymentAddressStr = options.deployerAddress;
                std::string bytecodeStr = options.Bytecode;

                size_t deploymentAddressLen = deploymentAddressStr.size() / 2;
                size_t bytecodeLen = bytecodeStr.size() / 2;

                unsigned char* deploymentAddress = new unsigned char[deploymentAddressLen];
                unsigned char* bytecode = new unsigned char[bytecodeLen];

                for (size_t i = 0; i < deploymentAddressLen; i++) {
                    deploymentAddress[i] = std::stoul(deploymentAddressStr.substr(i*2, 2), nullptr, 16);
                }

                for (size_t i = 0; i < bytecodeLen; i++) {
                    bytecode[i] = std::stoul(bytecodeStr.substr(i*2, 2), nullptr, 16);
                }

                size_t prefixLen = options.prefix.size() / 2;
                size_t suffixLen = options.suffix.size() / 2;

                unsigned char* prefix = new unsigned char[prefixLen];
                unsigned char* suffix = new unsigned char[suffixLen];

                for (size_t i = 0; i < prefixLen; i++) {
                    prefix[i] = std::stoul(options.prefix.substr(i*2, 2), nullptr, 16);
                }

                for (size_t i = 0; i < suffixLen; i++) {
                    suffix[i] = std::stoul(options.suffix.substr(i*2, 2), nullptr, 16);
                }

                unsigned char* d_deploymentAddress;
                unsigned char* d_bytecode;
                unsigned char* d_prefix;
                unsigned char* d_suffix;

                hipMalloc(&d_deploymentAddress, deploymentAddressLen * sizeof(unsigned char));
                checkCudaError(hipGetLastError(), "hipMalloc for d_deploymentAddress");
                hipMalloc(&d_bytecode, bytecodeLen * sizeof(unsigned char));
                checkCudaError(hipGetLastError(), "hipMalloc for d_bytecode");
                hipMalloc(&d_prefix, prefixLen * sizeof(unsigned char));
                checkCudaError(hipGetLastError(), "hipMalloc for d_prefix");
                hipMalloc(&d_suffix, suffixLen * sizeof(unsigned char));
                checkCudaError(hipGetLastError(), "hipMalloc for d_suffix");

                hipMemcpy(d_deploymentAddress, deploymentAddress, deploymentAddressLen * sizeof(unsigned char), hipMemcpyHostToDevice);
                checkCudaError(hipGetLastError(), "hipMemcpy from deploymentAddress to d_deploymentAddress");
                hipMemcpy(d_bytecode, bytecode, bytecodeLen * sizeof(unsigned char), hipMemcpyHostToDevice);
                checkCudaError(hipGetLastError(), "hipMemcpy from bytecode to d_bytecode");
                hipMemcpy(d_prefix, prefix, prefixLen * sizeof(unsigned char), hipMemcpyHostToDevice);
                checkCudaError(hipGetLastError(), "hipMemcpy from prefix to d_prefix");
                hipMemcpy(d_suffix, suffix, suffixLen * sizeof(unsigned char), hipMemcpyHostToDevice);
                checkCudaError(hipGetLastError(), "hipMemcpy from suffix to d_suffix");

                unsigned char* dev_validAddress;
                int* dev_validAddressesCount;
                hipMalloc((void**)&dev_validAddress, 20 * sizeof(unsigned char));
                checkCudaError(hipGetLastError(), "hipMalloc for dev_validAddress");
                hipMalloc((void**)&dev_validAddressesCount, sizeof(int));
                checkCudaError(hipGetLastError(), "hipMalloc for dev_validAddressesCount");

                hipMemset(dev_validAddressesCount, 0, sizeof(int));
                checkCudaError(hipGetLastError(), "hipMemset for dev_validAddressesCount");

                verifyContractAdresse<<<gridDim, blockDim>>>(d_deploymentAddress, deploymentAddressLen, d_bytecode, bytecodeLen,
                                                            dev_salts, numSalts, dev_validAddress, dev_validAddressesCount,
                                                            d_prefix, prefixLen, d_suffix, suffixLen);
                checkCudaError(hipGetLastError(), "verifyContractAdresse kernel launch");
                hipDeviceSynchronize();
                unsigned char* validAddress = new unsigned char[20];  
                int* validAddressesCount = new int(0);

                hipMemcpy(validAddress, dev_validAddress, 20 * sizeof(unsigned char), hipMemcpyDeviceToHost);
                checkCudaError(hipGetLastError(), "hipMemcpy from dev_validAddress to validAddress");

                hipMemcpy(validAddressesCount, dev_validAddressesCount, sizeof(int), hipMemcpyDeviceToHost);
                checkCudaError(hipGetLastError(), "hipMemcpy from dev_validAddressesCount to validAddressesCount");

                if (*validAddressesCount > 0) {
                    std::cout << "Valid address found:\n";
                    for (int i = 0; i < *validAddressesCount; ++i) {
                        std::cout << "Salt: ";
                        for (int j = 0; j < saltSize; ++j) {
                            std::cout << std::hex << std::setw(2) << std::setfill('0') << (int)h_salts[i * saltSize + j];
                        }
                        std::cout << std::dec << " Address: ";
                        for (int j = 0; j < 20; ++j) {
                            std::cout << std::hex << std::setw(2) << std::setfill('0') << (int)validAddress[j];
                        }
                        std::cout << std::dec << std::endl;
                    }
                }
                options.loopCount += 1;
                if (options.loop == true) {
                    if (options.loopCount % TRYTES_PER_TRIT == 0) {
                        std::cout << "Loop number: " << options.loopCount << std::endl;
                        bool decision = userprompt();
                        if (decision == true) {
                            continue;
                        } else {
                            break;
                        }
                    }
                } 
                else {
                    break;
                }

                delete[] validAddress;
                delete validAddressesCount;

                hipFree(dev_salts);
                hipFree(dev_validAddress);
                hipFree(dev_validAddressesCount);
                hipFree(devStates);

                delete[] deploymentAddress;
                delete[] bytecode;
                delete[] prefix;
                delete[] suffix;
            }

            delete[] h_salts;

            break;
        }






        default: {
            std::cout << "Invalid command. Valid commands are: account, contract" << std::endl;
            std::cout << "Use --help for more information.\n" << std::endl;
            return 1;
        }       
    }

    return 0;
}
